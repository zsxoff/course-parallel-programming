#include "hip/hip_runtime.h"
__global__ void addKernel(int *c, int *a, int *b) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  c[index] = a[index] + b[index];
}

#define kernel addKernel
#include "addGPU.c"
